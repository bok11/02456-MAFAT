#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

// Kernel declaration
__global__ void square_kernel_forward(float* __restrict__ output, 
                                      const float* __restrict__ input, 
                                      size_t N){
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i < N){

        output[i] = input[i] * input[i];
    }
    return;
}

__global__ void square_kernel_backward(float* __restrict__ output, 
                                       const float* __restrict__ input, 
                                       size_t N){
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i < N){
        output[i] = 2.0 * input[i];
    }
    return;
}

// Kernel launcher declaration
at::Tensor square_cuda_forward(at::Tensor input){
    const auto N = input.numel();
    auto output = at::zeros_like(input);    
    const int blockSize = 256;
    const int numBlocks = ceil((N + blockSize - 1) / blockSize);
    square_kernel_forward<<<numBlocks, blockSize>>>(output.data<float>(),
                                                    input.data<float>(),
                                                    N);
    return output;
}

at::Tensor square_cuda_backward(at::Tensor input){
    const auto N = input.numel();
    auto output = at::zeros_like(input);
    const int blockSize = 256;
    const int numBlocks = ceil((N + blockSize - 1) / blockSize);
	square_kernel_backward<<<numBlocks, blockSize>>>(output.data<float>(),
                                                     input.data<float>(),
                                                     N);
    return output;
}
